//nvcc -o libaddVecs.so addVecs.cu -shared -Xcompiler -fPIC -lgomp

#include <stdio.h>
#include <hip/hip_runtime.h>



__global__
void add_vecs(float *out, float *in1, float *in2, long n)
{
  //slightly more complicated version that should work with arbitrarily large arrays*/
  for (long i=threadIdx.x+blockDim.x*blockIdx.x;i<n;i+=gridDim.x*blockDim.x)
    out[i]=in1[i]+in2[i];
  
}

/*--------------------------------------------------------------------------------*/

__global__
void add_vecs_simple(float *out, float *in1, float *in2, long n)
{
  /* Simple way to add vectors where each element gets one thread.  works great
     for small arrays.*/
  long idx=threadIdx.x+blockDim.x*blockIdx.x;
  if (idx<n)
    out[idx]=in1[idx]+in2[idx];
}

/*--------------------------------------------------------------------------------*/
extern "C" {
void add(float *out, float *in1,float *in2, long n)
{  
  long bs=256;  //Set a block size for threads per block
  long nblock=n/bs;
  if ((nblock*bs)<n)
    nblock++;
  add_vecs_simple<<<nblock,bs>>>(out,in1,in2,n);
  printf("err is currently %s\n",hipGetErrorString(hipGetLastError()));

  if (1==0) {
    float *tmp=(float *)malloc(sizeof(float)*n);
    if (hipMemcpy(tmp,out,n*sizeof(float),hipMemcpyDeviceToHost)!=hipSuccess)
      printf("we had an issue with the memcpy.\n");
    else
      printf("first element of in1 is %f\n",tmp[0]);      
    free(tmp);
  }
  
}
}
/*--------------------------------------------------------------------------------*/
extern "C" {
void add2(float *out, float *in1,float *in2, long n)
{  
  long bs=256;
  long nblock=n/bs;
  if ((nblock*bs)<n)
    nblock++;
  long nblock_max=128;
  if (nblock>nblock_max)
    nblock=nblock_max;
  add_vecs<<<nblock,bs>>>(out,in1,in2,n);
  //printf("err is currently %s\n",cudaGetErrorString(cudaGetLastError()));

  if (1==0) {
    float *tmp=(float *)malloc(sizeof(float)*n);
    if (hipMemcpy(tmp,out,n*sizeof(float),hipMemcpyDeviceToHost)!=hipSuccess)
      printf("we had an issue with the memcpy.\n");
    else
      printf("first element of in1 is %f\n",tmp[0]);      
    free(tmp);
  }
  
}
}
/*--------------------------------------------------------------------------------*/
extern "C" {
void add3(float *out, float *in1,float *in2, long n,long nblock,long bs)
{
  //printf("block size/nblock are %ld %ld with n %ld\n",nblock,bs,n);
  add_vecs<<<nblock,bs>>>(out,in1,in2,n);
  //printf("err is currently %s\n",cudaGetErrorString(cudaGetLastError()));
  
}
}
