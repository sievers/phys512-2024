//nvcc -o liblaplace.so laplace_kernel.cu -shared -Xcompiler -fPIC -lgomp                             

#include <stdio.h>
#include <hip/hip_runtime.h>


__global__
void apply_stencil_cuda(float *out, float *in, long n, long m)
{
  for (long i=threadIdx.x+blockDim.x*blockIdx.x;i<n-1;i+=blockDim.x*gridDim.x)
    if (i>0) {
      for (long j=threadIdx.y+blockDim.y*blockIdx.y;j<m-1;j+=blockDim.y*gridDim.y)
	if (j>0) {
	  long ind=i*m+j;
	  float left=in[ind-1];
	  float right=in[ind+1];
	  float bot=in[ind-m];
	  float top=in[ind+m];
	  out[ind]=in[ind]-0.25*(left+right+top+bot);
	}
    }
}

/*--------------------------------------------------------------------------------*/
extern "C"
{
void apply_stencil(float *out, float *in, long n, long m)
{
  dim3 bs(16,16);
  dim3 nb(16,16);
  apply_stencil_cuda<<<nb,bs>>>(out,in,n,m);
  //printf("err is currently %s\n",cudaGetErrorString(cudaGetLastError()));
    
}
}
